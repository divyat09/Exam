
#include <hip/hip_runtime.h>
#include<stdio.h>
#include<stdlib.h>
#include<sys/time.h>

#define SEED 0x7457
#define NUM 10000000

#define CUDA_ERROR_EXIT(str) do{\
                                    hipError_t err = hipGetLastError();\
                                    if( err != hipSuccess){\
                                             printf("Cuda Error: '%s' for %s\n", hipGetErrorString(err), str);\
                                             exit(-1);\
                                    }\
                             }while(0);
#define TDIFF(start, end) ((end.tv_sec - start.tv_sec) * 1000000UL + (end.tv_usec - start.tv_usec))


__global__ void XOR(long long int *Data, int Size, int Odd)
{
      int tid = blockDim.x * blockIdx.x + threadIdx.x;
      int size = Size;
      int Bool= Odd;

      while(size!=0){
        if( tid < size )
            if( tid == size -1 && Bool == 1 ){
                // Do Nothing
            }
            else{
                Data[tid] = Data[tid] ^ Data[ tid + size ];
            }

            __syncthreads();
        
        // To avoid Infinite While Loop
        if (size==1)
        {
            return;
        }

        // Odd Number Case
        if( size % 2){

                size = size/2 +1;
                Bool = 1;
        }        
        else{

                Bool = 0;
                size = size / 2;    
        }
      }
}


int main(int argc, char **argv)
{
    struct timeval start, end, t_start, t_end;
    long long int *HArray;
    long long int *DArray;
    unsigned long num = NUM;   /*Default value of num from MACRO*/
    // int blocks;
    unsigned long Seed = SEED; /*Default value of Seed from MACRO*/

    if(argc == 3){
         num = atoi(argv[1]);   /*Update after checking*/
         if(num <= 0)
               num = NUM;
         
         Seed= atoi(argv[2]);
         if(Seed <= 0)
                Seed = SEED;
    }
    else{
        printf("%d", argc);
	    printf("Not Correct Number of Arguments");
        return -1;
    }


    /* Allocate host (CPU) memory and initialize*/

    HArray = (long long int*) malloc(num * sizeof(long long int) );
    if(!HArray){
          perror("malloc");
          exit(-1);
    }    

    srand(Seed);  
    for(int i=0;i<num;i++){
       HArray[i]= random();
    }

    for(int i=0;i<num;i++){
       printf("%lld ", HArray[i] );       
	if (i<num-1)
		printf("^ ");
    }

        
    gettimeofday(&t_start, NULL);
    
    /* Allocate GPU memory and copy from CPU --> GPU*/

    hipMalloc(&DArray, num * sizeof(long long int));
    CUDA_ERROR_EXIT("hipMalloc");

    hipMemcpy(DArray, HArray, num * sizeof(long long int) , hipMemcpyHostToDevice);
    CUDA_ERROR_EXIT("hipMemcpy");
    
    gettimeofday(&start, NULL);
    
    int blocks = num;
    
    if(num % 1024)
           ++blocks;

    // XOR<<<1, (num + num%2)/2>>>(DArray, num%2);

    if( num%2 ){
        XOR<<<blocks, 1024>>>(DArray, (num + 1)/2, 1);
        CUDA_ERROR_EXIT("kernel invocation");
    }
    else{
        XOR<<<blocks, 1024>>>(DArray, num/2, 0);
        CUDA_ERROR_EXIT("kernel invocation");
    }

    gettimeofday(&end, NULL);
    
    /* Copy back result*/

    hipMemcpy(HArray, DArray, num * sizeof(long long int) , hipMemcpyDeviceToHost);
    CUDA_ERROR_EXIT("memcpy");
    gettimeofday(&t_end, NULL);
    
    printf("\nTotal time = %ld microsecs Processsing =%ld microsecs\n", TDIFF(t_start, t_end), TDIFF(start, end));
    hipFree(DArray);
   
    /*Print the last element for sanity check*/ 
    printf("XOR: %lld\n", HArray[0]);
    
    free(HArray);
}
