
#include <hip/hip_runtime.h>
#include<stdio.h>
#include<stdlib.h>
#include<sys/time.h>

#define NUM 10000000

#define CUDA_ERROR_EXIT(str) do{\
                                    hipError_t err = hipGetLastError();\
                                    if( err != hipSuccess){\
                                             printf("Cuda Error: '%s' for %s\n", hipGetErrorString(err), str);\
                                             exit(-1);\
                                    }\
                             }while(0);
#define TDIFF(start, end) ((end.tv_sec - start.tv_sec) * 1000000UL + (end.tv_usec - start.tv_usec))

struct num_array{
                    double num1;
                    double num2;
                    double result;
};


__device__ void function(struct num_array *a)
{
    double square = a ->num1 * a->num1 +  a->num2 * a->num2  + 2 * a->num1 * a->num2;
    a->result = log(square)/sin(square);
    return;
}

__global__ void calculate(char *mem, int num)
{
      int i= blockIdx.x*blockDim.x*blockDim.y + blockDim.x*threadIdx.y + threadIdx.x;  
      if(i >= num)
           return;
       struct num_array *a = (struct num_array *)(mem + (i * 3 * sizeof(double)));
      function(a);
}

int main(int argc, char **argv)
{
    struct timeval start, end, t_start, t_end;
    int i;
    struct num_array *pa;
    char *ptr;
    char *sptr;
    char *gpu_mem;   
    unsigned long num = NUM;   /*Default value of num from MACRO*/
    int blocks;
    int NumRows, NumColumns;

    if(argc == 4){
         num = atoi(argv[1]);   /*Update after checking*/
         if(num <= 0)
               num = NUM;
         NumRows= atoi(argv[2]);
         NumColumns= atoi(argv[3]);
         
         if(NumRows==0 || NumColumns==0){
            printf("Error: Provide suitable values for Rows and Columns \n" );
            return -1;            
         }

         if(NumRows*NumColumns > 1024 ){
            printf("Error: Max thread limit per block exceeded \n" );
            return -1;
         }
    }
    else{
        printf("%d", argc);
	    printf("Not Correct Number of Argumnets");
        return -1;
    }

    /* Allocate host (CPU) memory and initialize*/

    ptr = (char *)malloc(num * 3 * sizeof(double));
    sptr = ptr; 
    for(i=0; i<num; ++i){
       pa = (struct num_array *) sptr;
       pa->num1 = (double) i + (double) i * 0.1;
       pa->num2 = pa->num1 + 1.0;
       sptr += 3 * sizeof(double);
    }
    
    
    gettimeofday(&t_start, NULL);
    
    /* Allocate GPU memory and copy from CPU --> GPU*/

    hipMalloc(&gpu_mem, num * 3 * sizeof(double));
    CUDA_ERROR_EXIT("hipMalloc");

    hipMemcpy(gpu_mem, ptr, num * 3 * sizeof(double) , hipMemcpyHostToDevice);
    CUDA_ERROR_EXIT("hipMemcpy");
    
    gettimeofday(&start, NULL);

    dim3 threads( NumRows , NumColumns ); 
    
    blocks = num /(NumRows*NumColumns);
    if(num % (NumRows*NumColumns))
           ++blocks;

    calculate<<<blocks, threads>>>(gpu_mem, num);
    CUDA_ERROR_EXIT("kernel invocation");
    gettimeofday(&end, NULL);
    
    /* Copy back result*/

    hipMemcpy(ptr, gpu_mem, num * 3 * sizeof(double) , hipMemcpyDeviceToHost);
    CUDA_ERROR_EXIT("memcpy");
    gettimeofday(&t_end, NULL);
    
    printf("Total time = %ld microsecs Processsing =%ld microsecs\n", TDIFF(t_start, t_end), TDIFF(start, end));
    hipFree(gpu_mem);
    sptr = ptr;
   
    /*Print the last element for sanity check*/ 
    pa = (struct num_array *) (sptr + (num -1)*3*sizeof(double));
    printf("num1=%f num2=%f result=%f\n", pa->num1, pa->num2, pa->result);

    
    free(ptr);
}
