
#include <hip/hip_runtime.h>
#include<stdio.h>
#include<stdlib.h>
#include<sys/time.h>


#define NUM 32

#define CUDA_ERROR_EXIT(str) do{\
                                    hipError_t err = hipGetLastError();\
                                    if( err != hipSuccess){\
                                             printf("Cuda Error: '%s' for %s\n", hipGetErrorString(err), str);\
                                             exit(-1);\
                                    }\
                             }while(0);


__global__ void D_Mul(int *dA, int *dB, int *dC)
{
      int i = threadIdx.x; 
  //    int i = blockIdx.x * blockDim.x + threadIdx.x;
      dC[i] = dA[i] * dB[i];
}

int main(int argc, char **argv)
{
    int ctr;
    int *hA, *hB, *hC;
  
    int *dA, *dB, *dC; 
   
    int size = NUM * sizeof(int); 

    /*Allocate memory on the host (CPU) */

    hA = (int *) malloc(size);
    if(!hA){
          perror("malloc");
          exit(-1);
    }
    
    hB = (int *) malloc(size);
    if(!hB){
          perror("malloc");
          exit(-1);
    }
       

    hC = (int *) malloc(size);
    if(!hC){
          perror("malloc");
          exit(-1);
    }
    
    /*Initialize hA and hB*/

    for(ctr=0; ctr < NUM; ++ctr)
         hA[ctr] = hB[ctr] = ctr+1;
    
   /*Allocate memory on the device (GPU) */

    hipMalloc(&dA,  size);
    CUDA_ERROR_EXIT("hipMalloc");

    hipMalloc(&dB,  size);
    CUDA_ERROR_EXIT("hipMalloc");

    hipMalloc(&dC,  size);
    CUDA_ERROR_EXIT("hipMalloc");
   
    /*Copy hA --> dA and hB --> dB */
    
    hipMemcpy(dA, hA, size, hipMemcpyHostToDevice);
    CUDA_ERROR_EXIT("memcpy1");
    
    hipMemcpy(dB, hB, size, hipMemcpyHostToDevice);
    CUDA_ERROR_EXIT("memcpy1");
    
    
     /*Invoke the kernel*/
     D_Mul<<<1, NUM>>>(dA, dB, dC);
    //int blocks = (NUM + 1023) >> 10;
    //D_Mul<<<blocks, 1024>>>(dA, dB, dC);
    CUDA_ERROR_EXIT("kernel invocation");
   
    printf("kernel successful\n"); 

    /*Copy back results*/
    hipMemcpy(hC, dC, size, hipMemcpyDeviceToHost);
    CUDA_ERROR_EXIT("memcpy");
    for(ctr=0; ctr < NUM; ++ctr)
        printf("%d %d %d\n", hA[ctr], hB[ctr], hC[ctr]);
    
    free(hA); 
    free(hB); 
    free(hC); 
    hipFree(dA);
    hipFree(dB);
    hipFree(dC);
}
